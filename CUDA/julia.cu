#include <stdio.h>
#include "julia.h"

#define DIM 800

struct hipComplex
{
	float r, i;

	hipComplex(float a, float b) : r(a), i(b) {}

	hipComplex operator*(const hipComplex &a) {
		return hipComplex(r * a.r, i * a.i);
	}

	hipComplex operator+(const hipComplex &a) {
		return hipComplex(r + a.r, i + a.i);
	}
};